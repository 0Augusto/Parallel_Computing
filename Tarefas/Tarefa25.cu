/*
 * @author(s) Filipe Arthur, Henrique Augusto Rodrigues
 * Matricula: XXXXXX, XXXXXX
 * Tarefa 25
  * @version 1.00
  */
 
/*
//-----------------PARCODE-----------------//
(Filipe)
*Tempo scan.c (Sequencial)

*real    0m0.401s
*user    0m0.188s
*sys     0m0.212s

*Tempo scan_incomplete_cuda.cu (Paralelo)

CUDA:

real    0m1.900s
user    0m0.830s
sys     0m0.996s

*SpeedUp = 0.401/1.900 = 0.21
//-----------------------------------------//

//-----------------MACBOOK Air (M1)-----------------//

(Henrique)
*Não há suporte para CUDA no Macbook Air (M1)

*Tivemos problemas em instalar a biblioteca <omp.h> no MacBook e o programa não pode ser executado

//===========Parallels VM UBUNTU 20.04 LTS=======//

*a[39999999] = 799999980000000.000000

*real	0m0.463s
*user	0m0.415s
*sys	0m0.048s



//===========Parallels VM UBUNTU 20.04 LTS=======//

//-----------------PARCODE-----------------//
(Henrique) scan.c
*a[39999999] = 799999980000000.000000

*real	0m0.640s
*user	0m0.394s
*sys	0m0.243s

scan_incomplete_cuda.cu

*a[39999999] = 799999980000000.000000

*real	0m2.175s
*user	0m1.098s
*sys	0m0.999s


SpeedUp = 0.640/2.175 = 0.294
//-----------------------------------------//
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void scan_cuda(double* a, double *s, int width) {
  // kernel scan
  int t = threadIdx.x;
    int b = blockIdx.x * blockDim.x;
    double x;
    // cria vetor na memória local
    __shared__ double p[1024];
    // carrega elementos do vetor da memória global para a local
    if (b + t < width)
        p[t] = a[b + t];
    // espera que todas as threads tenham carregado seus elementos
    __syncthreads();

    for (int i = 1; i < blockDim.x; i *= 2)
    {                            // realiza o scan em log n passos
        if (t >= i)              // verifica se a thread ainda participa neste passo
            x = p[t] + p[t - i]; // atribui a soma para uma variável temporária
        __syncthreads();         // espera threads fazerem as somas
        if (t >= i)
            p[t] = x; // copia a soma em definitivo para o vetor local
        __syncthreads();
    }
    if (b + t < width) // copia da memória local para a global
        a[b + t] = p[t];
    if (t == blockDim.x - 1)          // se for a última thread do bloco
        s[blockIdx.x + 1] = a[b + t]; // copia o seu valor para o vetor de saída
} 

__global__ void add_cuda(double *a, double *s, int width) {
  // kernel soma
  int t = threadIdx.x;
  int b = blockIdx.x * blockDim.x;
  // soma o somatório do último elemento do bloco anterior ao elemento atual
  if (b + t < width)
    a[b + t] += s[blockIdx.x];
}

int main()
{
  int width = 40000000;
  int size = width * sizeof(double);

  int block_size = 1024;
  int num_blocks = (width-1)/block_size+1;
  int s_size = (num_blocks * sizeof(double));  
 
  double *a = (double*) malloc (size);
  double *s = (double*) malloc (s_size);

  for(int i = 0; i < width; i++)
    a[i] = i;

  double *d_a, *d_s;

  // alocar vetores "a" e "s" no device
  hipMalloc((void **) &d_a, size);
  hipMalloc((void **) &d_s, s_size);

  // copiar vetor "a" para o device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

  // definição do número de blocos e threads (dimGrid e dimBlock)
  dim3 dimGrid(num_blocks,1,1);
  dim3 dimBlock(block_size,1,1);
  // chamada do kernel scan
  scan_cuda<<<dimGrid,dimBlock>>>(d_a, d_s, width);

  // copiar vetor "s" para o host
  hipMemcpy(s, d_s, s_size, hipMemcpyDeviceToHost);

  // scan no host (já implementado)
  s[0] = 0;
  for (int i = 1; i < num_blocks; i++)
    s[i] += s[i-1];
 
  // copiar vetor "s" para o device
  hipMemcpy(d_s, s, s_size, hipMemcpyHostToDevice);
  

  // chamada do kernel da soma
  add_cuda<<<dimGrid,dimBlock>>>(d_a, d_s, width);

  // copiar o vetor "a" para o host
  hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);

  printf("\na[%d] = %f\n",width-1,a[width-1]);
  
  hipFree(d_a);
  hipFree(d_s);
}
